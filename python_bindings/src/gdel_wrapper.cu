#include "hip/hip_runtime.h"
// CUDA wrapper to avoid C++ template symbol mangling issues
#include "gDel3D/GpuDelaunay.h"
#include "gDel3D/CommonTypes.h"

// C-style wrapper function to avoid template symbol mangling
extern "C"
{

    // Struct to hold point data in C-compatible format
    struct CPoint3
    {
        double x, y, z;
    };

    // Struct to hold output data in C-compatible format
    struct CDelaunayOutput
    {
        int *tetrahedra;
        int num_tetrahedra;
        double total_time;
        int success;
    };

    // C wrapper function that internally uses the C++ API
    int compute_delaunay_c(CPoint3 *points, int num_points, CDelaunayOutput *output)
    {
        try
        {
            // Convert C points to Point3HVec
            Point3HVec pointVec;
            pointVec.reserve(num_points);

            for (int i = 0; i < num_points; i++)
            {
                Point3 p;
                p._p[0] = points[i].x;
                p._p[1] = points[i].y;
                p._p[2] = points[i].z;
                pointVec.push_back(p);
            }

            // Create GpuDel instance and output
            GpuDel triangulator;
            GDelOutput gdelOutput;

            // Compute triangulation
            triangulator.compute(pointVec, &gdelOutput);

            // Convert output back to C format
            output->total_time = gdelOutput.stats.totalTime;
            output->success = 1;

            int nb_tetra_alive = 0;
            for (int i = 0; i < gdelOutput.tetInfoVec.size(); i++)
            {
                if (!isTetAlive(gdelOutput.tetInfoVec[i]))
                {
                    continue; // Skip dead tets
                }

                const Tet &tet = gdelOutput.tetVec[i];

                // Check all indices are differents
                if (tet._v[0] == tet._v[1] || tet._v[0] == tet._v[2] || tet._v[0] == tet._v[3] ||
                    tet._v[1] == tet._v[2] || tet._v[1] == tet._v[3] ||
                    tet._v[2] == tet._v[3])
                {
                    continue; // Skip invalid tetrahedra
                }

                // 32768 max
                if (tet._v[0] == num_points || tet._v[1] == num_points || tet._v[2] == num_points || tet._v[3] == num_points)
                {
                    continue; // Skip invalid tetrahedra
                }
                nb_tetra_alive++;
            }
            output->num_tetrahedra = nb_tetra_alive;

            // Allocate and copy tetrahedra data
            if (nb_tetra_alive > 0)
            {
                output->tetrahedra = new int[nb_tetra_alive * 4];
                int index = 0;
                for (int i = 0; i < gdelOutput.tetInfoVec.size(); i++)
                {
                    if (!isTetAlive(gdelOutput.tetInfoVec[i]))
                        continue; // Skip dead tets

                    const Tet &tet = gdelOutput.tetVec[i];
                    // Check all indices are different
                    if (tet._v[0] == tet._v[1] || tet._v[0] == tet._v[2] || tet._v[0] == tet._v[3] ||
                        tet._v[1] == tet._v[2] || tet._v[1] == tet._v[3] ||
                        tet._v[2] == tet._v[3])
                    {
                        continue; // Skip invalid tetrahedra
                    }

                    // 32768 max
                    if (tet._v[0] == num_points || tet._v[1] == num_points || tet._v[2] == num_points || tet._v[3] == num_points)
                    {
                        continue; // Skip invalid tetrahedra
                    }

                    output->tetrahedra[index * 4 + 0] = tet._v[0];
                    output->tetrahedra[index * 4 + 1] = tet._v[1];
                    output->tetrahedra[index * 4 + 2] = tet._v[2];
                    output->tetrahedra[index * 4 + 3] = tet._v[3];

                    index++;
                }
            }
            else
            {
                output->tetrahedra = nullptr;
            }

            return 1; // Success
        }
        catch (const std::exception &e)
        {
            output->success = 0;
            output->tetrahedra = nullptr;
            output->num_tetrahedra = 0;
            return 0; // Failure
        }
    }

    // Function to free allocated memory
    void free_delaunay_output_c(CDelaunayOutput *output)
    {
        if (output && output->tetrahedra)
        {
            delete[] output->tetrahedra;
            output->tetrahedra = nullptr;
        }
    }

} // extern "C"
